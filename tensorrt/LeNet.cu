#include "hip/hip_runtime.h"
#include "cudaUtility.h"
#include "opencv2/opencv.hpp"

__global__ void gpuPreLeNetNormalize(cv::cuda::PtrStepSzb input, float* output) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    output[y * 28 + x] = input(y, x) / 255.0;
}

hipError_t cudaPreLeNetNormalize(cv::cuda::GpuMat &input, float* output) {
    const dim3 blockDim(7, 7);
    const dim3 gridDim(4, 4);

    gpuPreLeNetNormalize<<<gridDim, blockDim>>>(input, output);

    return CUDA(hipGetLastError());
}
