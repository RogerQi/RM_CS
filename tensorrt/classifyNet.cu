#include "hip/hip_runtime.h"
#include "cudaUtility.h"
#include "opencv2/opencv.hpp"

__global__ void gpuImgCopy(cv::cuda::PtrStepSzb input, float* output, 
        uint32_t height, uint32_t width) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height)
        return ;

    output[y * 28 + x] = input(y, x);
}

hipError_t cudaImgCopy(cv::cuda::GpuMat &input, float* output, 
        uint32_t height, uint32_t width) {
    const dim3 blockDim(8, 8);
    const dim3 gridDim(iDivUp(width, 8), iDivUp(height, 8));

    gpuImgCopy<<<gridDim, blockDim>>>(input, output, height, width);

    return CUDA(hipGetLastError());
}
